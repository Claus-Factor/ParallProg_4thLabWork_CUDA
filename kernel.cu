﻿#include <iostream>
#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <vector>
#include <algorithm>
using namespace std;

// CUDA kernel для вычисления произведения в каждом блоке
__global__ void blockProductKernel(double* matrix, double* blockProducts, int rows, int cols) {
    __shared__ double partialProduct[1024];  // Shared память для частичных произведений
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;  // Глобальный индекс потока
    int tid = threadIdx.x;  // Локальный индекс потока в блоке

    // Проверяем, что поток не выходит за границы
    if (threadId < rows* cols) {
        partialProduct[tid] = matrix[threadId];  // Загружаем элемент в shared память
    }
    else {
        partialProduct[tid] = 1.0;  // Для потоков вне границ - нейтральное значение
    }
    __syncthreads();

    // Редукция произведения внутри блока
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            partialProduct[tid] *= partialProduct[tid + stride];
        }
        __syncthreads();
    }

    // Первый поток блока записывает результат редукции в массив блоков
    if (tid == 0) {
        blockProducts[blockIdx.x] = partialProduct[0];
    }
}


// Хелпер для проверки ошибок CUDA
void checkCuda(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    setlocale(LC_ALL, "rus");
    srand(static_cast<unsigned>(time(0)));

    const int rows = 10000;
    const int cols = rows;
    const int size = rows * cols;

    // Создание матрицы
    double* h_matrix = new double[size];

    // Заполнение матрицы случайными целыми числами от 1 до 10
    for (int i = 0; i < size; ++i) {
        h_matrix[i] = rand() % 10 + 1;  // Генерация целых чисел от 1 до 10
    }

    double* d_matrix;
    double* d_blockProducts;
    double h_result = 1.0;

    int threadsPerBlock = 256;

    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    double* h_blockProducts = new double[blocksPerGrid];

    // Выделение памяти на устройстве
    checkCuda(hipMalloc(&d_matrix, size * sizeof(double)), "Ошибка выделения памяти для d_matrix");
    checkCuda(hipMalloc(&d_blockProducts, blocksPerGrid * sizeof(double)), "Ошибка выделения памяти для d_blockProducts");
    checkCuda(hipMemcpy(d_matrix, h_matrix, size * sizeof(double), hipMemcpyHostToDevice), "Ошибка копирования матрицы на устройство");

    double min_time = 1e9;
    double max_time = 0.0;
    double total_time = 0.0;
    int checksCount = 3;

    for (int k = 0; k < checksCount; ++k) {
        double start_time = clock();

        // Запуск ядра для вычисления произведений в блоках
        blockProductKernel <<<blocksPerGrid, threadsPerBlock>>> (d_matrix, d_blockProducts, rows, cols);
        checkCuda(hipGetLastError(), "Ошибка запуска ядра");
        checkCuda(hipMemcpy(h_blockProducts, d_blockProducts, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost), "Ошибка копирования результатов блоков на хост");

        // Вычисление итогового произведения на хосте
        h_result = 1.0;  // Сброс результата перед новым измерением
        for (int i = 0; i < blocksPerGrid; ++i) {
            h_result *= h_blockProducts[i];
        }

        double run_time = (clock() - start_time) / CLOCKS_PER_SEC;

        if (run_time > max_time) max_time = run_time;
        if (run_time < min_time) min_time = run_time;
        total_time += run_time;
    }

    double average_time = total_time / checksCount;

    /*
    cout << "Матрица:" << endl;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            cout << h_matrix[i * cols + j] << " ";
        }
        cout << endl;
    }*/

    // Вывод результатов
    cout << "Произведение всех элементов матрицы: " << h_result << endl;
    cout << "Минимальное время: " << min_time << " секунд" << endl;
    cout << "Максимальное время: " << max_time << " секунд" << endl;
    cout << "Среднее время: " << average_time << " секунд\n" << endl;
        


    // Освобождение памяти
    hipFree(d_matrix);
    hipFree(d_blockProducts);
    delete[] h_matrix;
    delete[] h_blockProducts;


    return 0;

}




//int main() {
//    setlocale(LC_ALL, "rus");
//    srand(static_cast<unsigned>(time(0)));
//
//    const int rows = 32 * 192;
//    const int cols = rows;
//    const int size = rows * cols;
//
//    // Создание матрицы
//    double* h_matrix = new double[size];
//
//    // Заполнение матрицы случайными целыми числами от 1 до 10
//    for (int i = 0; i < size; ++i) {
//        h_matrix[i] = rand() % 10 + 1;  // Генерация целых чисел от 1 до 10
//    }
//
//    double* d_matrix;
//    double* d_blockProducts;
//    double h_result = 1.0;
//
//    //int threadsPerBlock = 160;
//    for (int threadsPerBlock = 32; threadsPerBlock <= 160; threadsPerBlock += 16) {
//
//        cout << threadsPerBlock << endl;
//
//        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
//
//        double* h_blockProducts = new double[blocksPerGrid];
//
//        // Выделение памяти на устройстве
//        checkCuda(cudaMalloc(&d_matrix, size * sizeof(double)), "Ошибка выделения памяти для d_matrix");
//        checkCuda(cudaMalloc(&d_blockProducts, blocksPerGrid * sizeof(double)), "Ошибка выделения памяти для d_blockProducts");
//        checkCuda(cudaMemcpy(d_matrix, h_matrix, size * sizeof(double), cudaMemcpyHostToDevice), "Ошибка копирования матрицы на устройство");
//
//        double min_time = 1e9;
//        double max_time = 0.0;
//        double total_time = 0.0;
//        int checksCount = 3;
//
//        for (int k = 0; k < checksCount; ++k) {
//            double start_time = clock();
//
//            // Запуск ядра для вычисления произведений в блоках
//            blockProductKernel << <blocksPerGrid, threadsPerBlock >> > (d_matrix, d_blockProducts, rows, cols);
//            checkCuda(cudaGetLastError(), "Ошибка запуска ядра");
//            checkCuda(cudaMemcpy(h_blockProducts, d_blockProducts, blocksPerGrid * sizeof(double), cudaMemcpyDeviceToHost), "Ошибка копирования результатов блоков на хост");
//
//            // Вычисление итогового произведения на хосте
//            h_result = 1.0;  // Сброс результата перед новым измерением
//            for (int i = 0; i < blocksPerGrid; ++i) {
//                h_result *= h_blockProducts[i];
//            }
//
//            double run_time = (clock() - start_time) / CLOCKS_PER_SEC;
//
//            if (run_time > max_time) max_time = run_time;
//            if (run_time < min_time) min_time = run_time;
//            total_time += run_time;
//        }
//
//        double average_time = total_time / checksCount;
//
//        /*
//        cout << "Матрица:" << endl;
//        for (int i = 0; i < rows; ++i) {
//            for (int j = 0; j < cols; ++j) {
//                cout << h_matrix[i * cols + j] << " ";
//            }
//            cout << endl;
//        }*/
//
//        // Вывод результатов
//        cout << "Произведение всех элементов матрицы: " << h_result << endl;
//        cout << "Минимальное время: " << min_time << " секунд" << endl;
//        cout << "Максимальное время: " << max_time << " секунд" << endl;
//        cout << "Среднее время: " << average_time << " секунд\n" << endl;
//        // Освобождение памяти
//
//        cudaFree(d_blockProducts);
//
//        delete[] h_blockProducts;
//    }
//
//    cudaFree(d_matrix);
//    delete[] h_matrix;
//
//
//    return 0;
//
//}



//
//int main() {
//    setlocale(LC_ALL, "rus");
//    srand(static_cast<unsigned>(time(0)));
//
//    const int rows = 32 * 192;
//    const int cols = rows;
//    const int size = rows * cols;
//
//    // Создание матрицы
//    double* h_matrix = new double[size];
//
//    // Заполнение матрицы случайными целыми числами от 1 до 10
//    for (int i = 0; i < size; ++i) {
//        h_matrix[i] = rand() % 10 + 1;  // Генерация целых чисел от 1 до 10
//    }
//
//    double* d_matrix;
//    double* d_blockProducts;
//    double h_result = 1.0;
//
//    // Массив для хранения времени для разных значений потоков
//    vector<double> times;
//
//    for (int threadsPerBlock = 32; threadsPerBlock <= 160; threadsPerBlock += 16) {
//
//        cout << "Тест с " << threadsPerBlock << " потоками на блок:\n";
//
//        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
//
//        double* h_blockProducts = new double[blocksPerGrid];
//
//        // Выделение памяти на устройстве
//        checkCuda(cudaMalloc(&d_matrix, size * sizeof(double)), "Ошибка выделения памяти для d_matrix");
//        checkCuda(cudaMalloc(&d_blockProducts, blocksPerGrid * sizeof(double)), "Ошибка выделения памяти для d_blockProducts");
//        checkCuda(cudaMemcpy(d_matrix, h_matrix, size * sizeof(double), cudaMemcpyHostToDevice), "Ошибка копирования матрицы на устройство");
//
//        double min_time = 1e9;
//        double max_time = 0.0;
//        double total_time = 0.0;
//        int checksCount = 3;
//
//        for (int k = 0; k < checksCount; ++k) {
//            double start_time = clock();
//
//            // Запуск ядра для вычисления произведений в блоках
//            blockProductKernel << <blocksPerGrid, threadsPerBlock >> > (d_matrix, d_blockProducts, rows, cols);
//            checkCuda(cudaGetLastError(), "Ошибка запуска ядра");
//            checkCuda(cudaMemcpy(h_blockProducts, d_blockProducts, blocksPerGrid * sizeof(double), cudaMemcpyDeviceToHost), "Ошибка копирования результатов блоков на хост");
//
//            // Вычисление итогового произведения на хосте
//            h_result = 1.0;  // Сброс результата перед новым измерением
//            for (int i = 0; i < blocksPerGrid; ++i) {
//                h_result *= h_blockProducts[i];
//            }
//
//            double run_time = (clock() - start_time) / CLOCKS_PER_SEC;
//
//            if (run_time > max_time) max_time = run_time;
//            if (run_time < min_time) min_time = run_time;
//            total_time += run_time;
//        }
//
//        double average_time = total_time / checksCount;
//        times.push_back(average_time);  // Добавляем среднее время в массив
//
//        cout << "Произведение всех элементов матрицы: " << h_result << endl;
//        cout << "Минимальное время: " << min_time << " секунд" << endl;
//        cout << "Максимальное время: " << max_time << " секунд" << endl;
//        cout << "Среднее время: " << average_time << " секунд\n" << endl;
//
//        delete[] h_blockProducts;
//    }
//
//    // Освобождение памяти
//    cudaFree(d_matrix);
//    cudaFree(d_blockProducts);
//    delete[] h_matrix;
//
//    // Построение графика зависимости времени выполнения от количества потоков
//    cout << "\nГрафик зависимости времени выполнения от количества потоков:\n";
//
//    // Определяем масштаб для вывода графика
//    double max_time_for_graph = *std::max_element(times.begin(), times.end());
//    int graph_width = 50;
//
//    for (int i = 0; i < times.size(); ++i) {
//        int bar_length = static_cast<int>((times[i] / max_time_for_graph) * graph_width);
//        cout << (32 + i * 32) << " поток(ов): " << string(bar_length, '*') << " " << times[i] << " секунд\n";
//    }
//
//    return 0;
//}
//

